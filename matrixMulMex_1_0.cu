#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
// Function name:   matrixMulMexCUBLAS_streams_2
// Description:     -Function will multiply matrix_a with matrix_b using the
//
// Input:           l_matrix(M x K x mat_num), r_matrix(K x N x mat_num),stream_num)
// Return Value:    outMatrix(M x N x mat_num)
////////////////////////////////////////////////////////////////////////////

// Utilities and system includes
#include <assert.h>
#include <windows.h>
#include "mex.h"
#include "blas.h"
#include <cublasXt.h>


#if !defined(_WIN32)
#define sgemm sgemm_
#endif


// CUDA runtime
#include <stdio.h> 
#include <stdlib.h> 
#include <math.h> 

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
//#include "hipblas.h"
#include <pthread.h>
#include <DMATransfer.h>

// #include "cblas.h"

// global variables
int thread_num 		= 0;
float fpga_ratio 	= 0.0;
float cpu_ratio 	= 1.0;

typedef struct _thread_data_t {
	int tid;
// 	enum CBLAS_ORDER Order;
    char* TransA;
    char* TransB; 
    ptrdiff_t M;
    ptrdiff_t N;
    ptrdiff_t K; 
    float alpha;
    float *A;
    ptrdiff_t lda;
    float *B;
    ptrdiff_t ldb;
    float beta;
    float *C;
    ptrdiff_t ldc;
} thread_data_t;

typedef struct _matrixSize {    // Optional Command-line multiplier for matrix sizes
    unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;
} sMatrixSize;


void* run_cpu_blas(void *threadarg) {
    thread_data_t *d = (thread_data_t*)threadarg;
    ptrdiff_t *m = &(d->M);
    ptrdiff_t *n = &(d->N);
    ptrdiff_t *k = &(d->K);
    ptrdiff_t *LDA = &(d->lda);
    ptrdiff_t *LDB = &(d->ldb);
    ptrdiff_t *LDC = &(d->ldc);
    sgemm( 
        d->TransA, 
        d->TransA, 
        m, 
        n, 
        k, 
        &(d->alpha), 
        d->A, 
        LDA, 
        d->B, 
        LDB, 
        &(d->beta), 
        d->C, 
        LDC);   
    //printf("finished cpu calc\n");
    pthread_exit(NULL);
    return NULL;
}


void* run_fpga_blas(void *threadarg) {
    // DWORD BUF_SIZE = 536870912;
	// UCHAR* baseWriteBuffer = new UCHAR[BUF_SIZE];
	// UCHAR* baseReadBuffer = new UCHAR[BUF_SIZE];
    // DMATransfer(baseWriteBuffer, BUF_SIZE, baseReadBuffer, BUF_SIZE);
    // delete[] baseWriteBuffer;
	// delete[] baseReadBuffer;
    thread_data_t *d = (thread_data_t*)threadarg;
    
    
    ptrdiff_t *m = &(d->M);
    ptrdiff_t *n = &(d->N);
    ptrdiff_t *k = &(d->K);
    ptrdiff_t *LDA = &(d->lda);
    ptrdiff_t *LDB = &(d->ldb);
    ptrdiff_t *LDC = &(d->ldc);
    sgemm( 
        d->TransA, 
        d->TransA, 
        m, 
        n, 
        k, 
        &(d->alpha), 
        d->A, 
        LDA, 
        d->B, 
        LDB, 
        &(d->beta), 
        d->C, 
        LDC);
    return NULL;
}

void cblas_sgemm_wrapper( 
    char *transa, 
    char *transb, 
    int  *m_p, 
    int  *n_p, 
    int  *k_p, 
    float *alpha_p, 
    float *h_A, 
    int *lda_p, 
    float *h_B, 
    int *ldb_p, 
    float *beta_p, 
    float *h_C, 
    int *ldc_p) {               


    ptrdiff_t m,n,k;
    m = *m_p;       
    n = *n_p;
    k = *k_p;
    ptrdiff_t lda,ldb,ldc;
    lda = *lda_p;
    ldb = *ldb_p;
    ldc = *ldc_p;
    
    float alpha = *alpha_p;
    float beta = *beta_p;

	// set up the division between cpu and fpga
	float cpu_inner_ratio 	= cpu_ratio / (fpga_ratio + cpu_ratio);
	float fpga_inner_ratio 	= 1.0 - cpu_inner_ratio;
	ptrdiff_t eff_fpga_n_all	= (ptrdiff_t) ((float)fpga_inner_ratio * (float)n);
    ptrdiff_t eff_fpga_n 		= (thread_num == 1)     ? 0 : (ptrdiff_t) (ptrdiff_t)eff_fpga_n_all / (ptrdiff_t)(thread_num - 1);
	ptrdiff_t eff_cpu_n 		= (eff_fpga_n_all == 0) ? n : (ptrdiff_t) (n - eff_fpga_n_all) + (ptrdiff_t) (eff_fpga_n_all % (ptrdiff_t)(thread_num - 1));	
    
	pthread_t* threads = (pthread_t*)mxMalloc(sizeof(pthread_t)*thread_num);
    if (threads==NULL){
        mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Out of system memory for threads.");
        return;
    }
    
    thread_data_t* thr_data = (thread_data_t*)mxMalloc(sizeof(thread_data_t)*thread_num);
    if (thr_data==NULL){
        mxFree(threads);
        mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Out of system memory for thread structure.");
        return;
    }
    
	for (ptrdiff_t t = 0; t < (ptrdiff_t)thread_num; t++) {//set data for pthread
        thr_data[t].tid     = t;	
        thr_data[t].TransA 	= transa;
        thr_data[t].TransB 	= transb;
        thr_data[t].M 		= m;
		// divide matricies and give the leftover to the last thread
        thr_data[t].N		= (t == (size_t)thread_num - 1) ? eff_cpu_n :
															  eff_fpga_n;
        thr_data[t].K 		= k;
        thr_data[t].alpha	= alpha;
        thr_data[t].A		= h_A ;
        thr_data[t].lda		= lda;
        thr_data[t].B		= (h_B + (int)((ptrdiff_t)t * (ptrdiff_t)ldb * (ptrdiff_t)eff_fpga_n));
        thr_data[t].ldb		= ldb;
        thr_data[t].beta	= beta;
        thr_data[t].C		= (h_C + (int)((ptrdiff_t)t * (ptrdiff_t)ldc * (ptrdiff_t)eff_fpga_n));
        thr_data[t].ldc		= ldc;
    }
    
	int rc; //error code for thread createion 
    for (int t = 0; t < thread_num; t++) {
        rc = (t == thread_num - 1) ? pthread_create(&threads[t], NULL, run_cpu_blas, &thr_data[t]) : 
                                     pthread_create(&threads[t], NULL, run_fpga_blas, &thr_data[t]);
        if (rc) {
            mxFree(threads);
            mxFree(thr_data);            
            mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Bad return code for pthread_create");
            return;
        }
    }
     
	for (int t = 0; t < thread_num; t++) {
		pthread_join(threads[t], NULL);
	}

    mxFree(threads);
    mxFree(thr_data);
    return;        
}

// function to create an int aray of on system gpus (NVIDIA only)
int* InitiateGpuId(int *gpu_num){
	//initilize variables
    int gpu_max_num     = 0;
    hipError_t cuda_error_check;
    
	// get devices number
    cuda_error_check    = hipGetDeviceCount(&gpu_max_num);
    if (cuda_error_check != hipSuccess || gpu_max_num == 0){
        mexWarnMsgTxt ("No active GPU on system.");
        return NULL;
    }
    
	// assign memory for the gpu numbers array
    int *gpu_id = (int*)mxMalloc(sizeof(int)*gpu_max_num);
    if (gpu_id==NULL){
        mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Out of system memory for gpu array.");
        return NULL;
    }
	
	// assign gpu ids to the array
	for(int i = 0; i < gpu_max_num; i++){
        gpu_id[i] = i;
    }
    gpu_max_num = *gpu_num < gpu_max_num ? *gpu_num : gpu_max_num;
	*gpu_num    = (gpu_max_num == 0) ? 1 : gpu_max_num;
    for(int i = 0; i < gpu_max_num; i++){
        gpu_id[i] = i;
    }
	
    return gpu_id;
}

int initializeCUDA(int &devID, int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[], sMatrixSize &matrix_size)
{
    matrix_size.uiWA = *(mxGetDimensions(prhs[0]) + 1);
    matrix_size.uiHA = *(mxGetDimensions(prhs[0]) + 0);
    matrix_size.uiWB = *(mxGetDimensions(prhs[1]) + 1);
    matrix_size.uiHB = *(mxGetDimensions(prhs[1]) + 0);
    matrix_size.uiWC = *(mxGetDimensions(prhs[1]) + 1);
    matrix_size.uiHC = *(mxGetDimensions(prhs[0]) + 0);
    return 0;
}

int matrixMultiply_wrapper(float* h_A, float* h_B, float* h_C, int devID, sMatrixSize &matrix_size, int gpu_num, int block_dim)
{
	// initialize variables
    hipblasStatus_t  cublas_error_check; // error checker
    float alpha = 1.0f;					// constant for BLAS routine
    float beta  = 0.0f;					// constant for BLAS routine
    cublasXtHandle_t handle;			// handle for cuBLAS_XT routine
	int *gpu_id = NULL;					// gpu array DIM	
	
    if (HIPBLAS_STATUS_SUCCESS != (cublasXtCreate(&handle))){
        mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Could not create CUBLAS handle.");
        return 1;
    }    
    
	// create an INT array of the gpus in system (NVIDIA only)
    gpu_id = InitiateGpuId(&gpu_num);
    if (gpu_id == NULL && gpu_num != 0){
		mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Could not set up available gpus array.");
		return 1;
	}

	// create xt handle, this is data for the XT routine
    cublas_error_check = cublasXtDeviceSelect(handle, gpu_num, gpu_id);
    if (cublas_error_check != HIPBLAS_STATUS_SUCCESS){
        mxFree(gpu_id);
		mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Could not set gpu device for cuBLAS_XT.");
        return 0;
    }     

	// set the cpu routine
    cublas_error_check = cublasXtSetCpuRoutine(handle, CUBLASXT_GEMM, CUBLASXT_FLOAT, cblas_sgemm_wrapper);
    if (cublas_error_check != HIPBLAS_STATUS_SUCCESS){
		mxFree(gpu_id);
		mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Cannot set cpu and fpga routine.");        
        return 0;
    }
    
	// set CPU and fpga routine ratio
    cublas_error_check = cublasXtSetCpuRatio(handle, CUBLASXT_GEMM, CUBLASXT_FLOAT, cpu_ratio + fpga_ratio);   
    if (cublas_error_check != HIPBLAS_STATUS_SUCCESS){
		mxFree(gpu_id);	
		mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Cannot set cpu and fpga offload.");        
        return 0;
    } 
    
	// set clock dim if recieved
	if (block_dim != 0){
		cublas_error_check = cublasXtSetBlockDim(handle , block_dim);
		if (cublas_error_check != HIPBLAS_STATUS_SUCCESS){
			mxFree(gpu_id);	
			mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Cannot set cpu and fpga offload.");        
        return 0;
    } 
	}
	
	// pin the memory
    cublas_error_check = cublasXtSetPinningMemMode(handle,CUBLASXT_PINNING_ENABLED);
        if (cublas_error_check != HIPBLAS_STATUS_SUCCESS){
        mxFree(gpu_id);
		mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Cannot pin memory.");        
        return 0;
    } 
	
	// call cuBLAS_XT routine to perform the multipictaion (matricies shoud be in COLUMN MAJOR FORMAT)
    cublas_error_check = (cublasXtSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                      matrix_size.uiHA, matrix_size.uiWB, matrix_size.uiWA, &alpha,
                                      (h_A), matrix_size.uiHA,
                                      (h_B), matrix_size.uiHB,
                                      &beta,
                                      (h_C), matrix_size.uiHA));
    if (cublas_error_check != HIPBLAS_STATUS_SUCCESS){
        mxFree(gpu_id);
		mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Could not run device CUBLAS_XT kernal.");        
        return 0;
    }


	// Destroy the handle.
    cublas_error_check = (cublasXtDestroy(handle));
    if (cublas_error_check != HIPBLAS_STATUS_SUCCESS){
		mxFree(gpu_id);
		mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Could not destroy CUBLAS handle.");        
        return 0;
    }    
    mxFree(gpu_id);
    return 1;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
/* The gateway function */
void mexFunction( int nlhs, mxArray *plhs[],
                  int nrhs, const mxArray *prhs[])	
{
    
    int devID = 0;
    sMatrixSize matrix_size;

	// check for proper number of arguments
    if(nrhs > 7 || nrhs < 2) {
        mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Bad input.");
    }
    if(nlhs!=1) {
        mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","One output required.");
    }
	
	// Mex reading arguments
    float* l_matrix;               	// left matrix
	float* r_matrix;               	// right matrix
	float* outMatrix;              	// output matrix
    cpu_ratio  			= 1.0;      // cpu usage ratio, global variable
	fpga_ratio 			= 0.0;		// fpga useage raio, global variable
	int block_dim 		= 0;		// block dimensions for cuBLAS_XT algorithem
    int gpu_num 		= 0;        // number of GPU to use with XT    
	thread_num			= 1; 		// thread number, global variable
	
	// Assign inputs to corresponding variables
	switch ( nrhs ) {
		case 7:
			block_dim 		= *(int*)mxGetData(prhs[6]);
		case 6:			
			fpga_ratio		= *(float*)mxGetData(prhs[5]);
		case 5:
			cpu_ratio       = *(float*)mxGetData(prhs[4]);
			if (fpga_ratio + cpu_ratio > 1.0){
				mexWarnMsgTxt("matrixMulMex_1_0 : Fpga and cpu ratio inputs exceed 1.0, using 1.0 cpu instead.");
				fpga_ratio 	= 0.0; //glogal variable 
				cpu_ratio	= 1.0;
			}
		case 4:
			gpu_num         = *(int*)mxGetData(prhs[3]);			
		case 3:
			thread_num      = *(int*)mxGetData(prhs[2]) == 0 ? 1 : 
                                                               *(int*)mxGetData(prhs[2]); //glogal variable 
            if (fpga_ratio != 0 && cpu_ratio != 0 && thread_num < 2){
				mexWarnMsgTxt("matrixMulMex_1_0 : not enough threads inputted, using one thread instead, cpu only.");
				fpga_ratio 	= 0.0; //glogal variable 
				cpu_ratio	= 1.0;
                thread_num  = 1;
			}
		default:
			r_matrix        = (float*)mxGetData(prhs[1]);
			l_matrix        = (float*)mxGetData(prhs[0]);			
			break;
	}

    // check that number of rows in matricies are legal 
    if(*(mxGetDimensions(prhs[0]) + 1) != *(mxGetDimensions(prhs[1]) + 0)) {
        mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Input must be legal.");
    }
    
	// build a struct with the dimensions of all the matricies
    if (1 == initializeCUDA(devID, nlhs, plhs, nrhs, prhs, matrix_size)){
		mexErrMsgIdAndTxt("MyToolbox:arrayProduct:notRowVector","Unknown issue with matricies.");
		return;
	}
    
	// create memory fo the left hand side result
	mwSize dims[2];	
	dims[0] = matrix_size.uiHA;
	dims[1] = matrix_size.uiWB;
    plhs[0] = mxCreateNumericArray(2, dims, mxSINGLE_CLASS, mxREAL);
    
    // get a pointer to the real data in the output matrix.
    outMatrix = (float*)mxGetData(plhs[0]);

	// call the cuBLAS_XT wrapper function.
    int matrix_result = matrixMultiply_wrapper(	l_matrix, 
												r_matrix, 
												outMatrix, 
												devID,
												matrix_size, 
												gpu_num, 
												block_dim
												);	
	// initzilize the global variables
	cpu_ratio  = 1.0; //global variable
	fpga_ratio = 0.0; //global variable 
    thread_num = 0; //global variable 
    return;
}
